#include "hip/hip_runtime.h"
#define N 12
#define SCOUNT PCOUNT

__constant__ uint32_t pow2[9] = {1, 2, 4, 8, 16, 32, 64, 128, 256};

__constant__ uint32_t binvert_limb_table[128] = {
  0x01, 0xAB, 0xCD, 0xB7, 0x39, 0xA3, 0xC5, 0xEF,
  0xF1, 0x1B, 0x3D, 0xA7, 0x29, 0x13, 0x35, 0xDF,
  0xE1, 0x8B, 0xAD, 0x97, 0x19, 0x83, 0xA5, 0xCF,
  0xD1, 0xFB, 0x1D, 0x87, 0x09, 0xF3, 0x15, 0xBF,
  0xC1, 0x6B, 0x8D, 0x77, 0xF9, 0x63, 0x85, 0xAF,
  0xB1, 0xDB, 0xFD, 0x67, 0xE9, 0xD3, 0xF5, 0x9F,
  0xA1, 0x4B, 0x6D, 0x57, 0xD9, 0x43, 0x65, 0x8F,
  0x91, 0xBB, 0xDD, 0x47, 0xC9, 0xB3, 0xD5, 0x7F,
  0x81, 0x2B, 0x4D, 0x37, 0xB9, 0x23, 0x45, 0x6F,
  0x71, 0x9B, 0xBD, 0x27, 0xA9, 0x93, 0xB5, 0x5F,
  0x61, 0x0B, 0x2D, 0x17, 0x99, 0x03, 0x25, 0x4F,
  0x51, 0x7B, 0x9D, 0x07, 0x89, 0x73, 0x95, 0x3F,
  0x41, 0xEB, 0x0D, 0xF7, 0x79, 0xE3, 0x05, 0x2F,
  0x31, 0x5B, 0x7D, 0xE7, 0x69, 0x53, 0x75, 0x1F,
  0x21, 0xCB, 0xED, 0xD7, 0x59, 0xC3, 0xE5, 0x0F,
  0x11, 0x3B, 0x5D, 0xC7, 0x49, 0x33, 0x55, 0xFF
};


typedef struct {
  uint32_t index;
  uint32_t hashid;
  uint8_t origin;
  uint8_t chainpos;
  uint8_t type;
  uint8_t reserved;
} fermat_t;

typedef struct {
  uint32_t N_;
  uint32_t SIZE_;
  uint32_t STRIPES_;
  uint32_t WIDTH_;
  uint32_t PCOUNT_;
  uint32_t TARGET_;
  uint32_t LIMIT13_;
  uint32_t LIMIT14_;
  uint32_t LIMIT15_;
} config_t;

__global__ void getconfig(config_t *conf)
{
  config_t c;
  c.N_ = N;
  c.SIZE_ = SIZE;
  c.STRIPES_ = STRIPES;
  c.WIDTH_ = WIDTH;
  c.PCOUNT_ = PCOUNT;
  c.TARGET_ = TARGET;
  c.LIMIT13_ = LIMIT13;
  c.LIMIT14_ = LIMIT14;
  c.LIMIT15_ = LIMIT15;
  *conf = c;
}

__device__ void shl32(uint32_t *data, unsigned size)
{
  #pragma unroll
  for (int j = size-1; j > 0; j--)
    data[j] = data[j-1];
  data[0] = 0;
}

__device__ void shr32(uint32_t *data, unsigned size)
{
  #pragma unroll
  for (int j = 1; j < size; j++)
    data[j-1] = data[j];
  data[size-1] = 0;
}

__device__ void shl(uint32_t *data, unsigned size, unsigned bits)
{
  #pragma unroll
  for(int i = size-1; i > 0; i--)
    data[i] = (data[i] << bits) | (data[i-1] >> (32-bits));
  
  data[0] = data[0] << bits;
}

__device__ void shr(uint32_t *data, unsigned size, unsigned bits)
{
  #pragma unroll
  for(int i = 0; i < size-1; i++)
    data[i] = (data[i] >> bits) | (data[i+1] << (32-bits));
  data[size-1] = data[size-1] >> bits;
}

__device__ void shlreg(uint32_t *data, unsigned size, unsigned bits)
{
  for (unsigned i = 0, ie = bits/32; i < ie; i++)
    shl32(data, size);
  
  if (bits%32)
    shl(data, size, bits%32);
}


__device__ void shrreg(uint32_t *data, unsigned size, unsigned bits)
{
  for (unsigned i = 0, ie = bits/32; i < ie; i++)
    shr32(data, size);
  if (bits%32)
    shr(data, size, bits%32);
}

__device__ uint32_t add128(uint4 *A, uint4 B)
{
//   *A += B; 
  A->x += B.x;
  A->y += B.y;
  A->z += B.z;
  A->w += B.w;
//   uint4 carry = -convert_uint4((*A) < B);
  uint4 carry = { -(A->x < B.x), -(A->y < B.y), -(A->z < B.z), -(A->w < B.w) };
  
  (*A).y += carry.x; carry.y += ((*A).y < carry.x);
  (*A).z += carry.y; carry.z += ((*A).z < carry.y);
  (*A).w += carry.z;
  return carry.w + ((*A).w < carry.z); 
}


__device__ uint32_t add128Carry(uint4 *A, uint4 B, uint32_t externalCarry)
{
//   *A += B;
  A->x += B.x;
  A->y += B.y;
  A->z += B.z;
  A->w += B.w;  
//   uint4 carry = -convert_uint4((*A) < B);
  uint4 carry = { -(A->x < B.x), -(A->y < B.y), -(A->z < B.z), -(A->w < B.w) };
  
  (*A).x += externalCarry; carry.x += ((*A).x < externalCarry);
  (*A).y += carry.x; carry.y += ((*A).y < carry.x);
  (*A).z += carry.y; carry.z += ((*A).z < carry.y);
  (*A).w += carry.z;
  return carry.w + ((*A).w < carry.z); 
}

__device__ uint32_t add256(uint4 *a0, uint4 *a1, uint4 b0, uint4 b1)
{
  return add128Carry(a1, b1, add128(a0, b0));
}

__device__ uint32_t add384(uint4 *a0, uint4 *a1, uint4 *a2, uint4 b0, uint4 b1, uint4 b2)
{
  return add128Carry(a2, b2, add128Carry(a1, b1, add128(a0, b0)));
}

__device__ uint32_t add512(uint4 *a0, uint4 *a1, uint4 *a2, uint4 *a3, uint4 b0, uint4 b1, uint4 b2, uint4 b3)
{
  return add128Carry(a3, b3, add128Carry(a2, b2, add128Carry(a1, b1, add128(a0, b0))));
}

__device__ uint32_t sub64Borrow(uint2 *A, uint2 B, uint32_t externalBorrow)
{
//   uint2 borrow = -convert_uint2((*A) < B);
  uint2 borrow = { -(A->x < B.x), -(A->y < B.y) };
//   *A -= B;
  A->x -= B.x;
  A->y -= B.y;
  
  borrow.x += (*A).x < externalBorrow; (*A).x -= externalBorrow;
  borrow.y += (*A).y < borrow.x; (*A).y -= borrow.x;
  return borrow.y;
}

__device__ uint32_t sub96Borrow(uint4 *A, uint4 B, uint32_t externalBorrow)
{
  //   uint2 borrow = -convert_uint2((*A) < B);
  uint4 borrow = {
    (*A).x < B.x,
      (*A).y < B.y,
      (*A).z < B.z,
      0
  };
  (*A).x -= B.x;
  (*A).y -= B.y;
  (*A).z -= B.z;
  
  borrow.x += (*A).x < externalBorrow; (*A).x -= externalBorrow;
  borrow.y += (*A).y < borrow.x; (*A).y -= borrow.x;
  borrow.z += (*A).z < borrow.y; (*A).z -= borrow.y;
  
  return borrow.z;
}

__device__ uint32_t sub128(uint4 *A, uint4 B)
{
  uint4 borrow = {
    (*A).x < B.x,
      (*A).y < B.y,
      (*A).z < B.z,
      (*A).w < B.w
  };
  (*A).x -= B.x;
  (*A).y -= B.y;
  (*A).z -= B.z;
  (*A).w -= B.w;  
  
  borrow.y += (*A).y < borrow.x; (*A).y -= borrow.x;
  borrow.z += (*A).z < borrow.y; (*A).z -= borrow.y;
  borrow.w += (*A).w < borrow.z; (*A).w -= borrow.z;
  return borrow.w;
}

__device__ uint32_t sub128Borrow(uint4 *A, uint4 B, uint32_t externalBorrow)
{
//   uint4 borrow = -convert_uint4((*A) < B);
  uint4 borrow = { -(A->x < B.x), -(A->y < B.y), -(A->z < B.z), -(A->w < B.w) };  
//   *A -= B;
  A->x -= B.x;
  A->y -= B.y;
  A->z -= B.z;
  A->w -= B.w;  
  
  borrow.x += (*A).x < externalBorrow; (*A).x -= externalBorrow;
  borrow.y += (*A).y < borrow.x; (*A).y -= borrow.x;
  borrow.z += (*A).z < borrow.y; (*A).z -= borrow.y;
  borrow.w += (*A).w < borrow.z; (*A).w -= borrow.z;
  return borrow.w;
}

__device__ uint32_t sub256(uint4 *a0, uint4 *a1, uint4 b0, uint4 b1)
{
  return sub128Borrow(a1, b1, sub128(a0, b0));
}

__device__ uint32_t sub320(uint4 *a0, uint4 *a1, uint2 *a2, uint4 b0, uint4 b1, uint2 b2)
{
  return sub64Borrow(a2, b2, sub128Borrow(a1, b1, sub128(a0, b0)));
}

__device__ uint32_t sub352(uint4 *a0, uint4 *a1, uint4 *a2, uint4 b0, uint4 b1, uint4 b2)
{
  return sub96Borrow(a2, b2, sub128Borrow(a1, b1, sub128(a0, b0)));
}

__device__ uint32_t sub384(uint4 *a0, uint4 *a1, uint4 *a2, uint4 b0, uint4 b1, uint4 b2)
{
  return sub128Borrow(a2, b2, sub128Borrow(a1, b1, sub128(a0, b0)));
}

__device__ uint32_t sub448(uint4 *a0, uint4 *a1, uint4 *a2, uint2 *a3, uint4 b0, uint4 b1, uint4 b2, uint2 b3)
{
  return sub64Borrow(a3, b3, sub128Borrow(a2, b2, sub128Borrow(a1, b1, sub128(a0, b0))));
}

__device__ uint32_t invert_limb(uint32_t limb)
{
  uint32_t inv = binvert_limb_table[(limb/2) & 0x7F];
  inv = 2*inv - inv*inv*limb;
  inv = 2*inv - inv*inv*limb;
  return -inv;
}
